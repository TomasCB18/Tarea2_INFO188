#include <iostream>
#include <vector>
#include <cstdlib>
#include <random>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <ctime>
#include <fstream>
#include <sys/stat.h>
#include <sys/types.h>

using namespace std;

// fun para inicializar un arreglo con valores aleatorios
void initializeArray(int *arr, int n) {
    mt19937 rng(static_cast<unsigned int>(time(0)));
    uniform_int_distribution<int> dist(0, 999999);
    for (int i = 0; i < n; ++i) {
        arr[i] = dist(rng);
    }
}

// merge-Sort paralelo para CPU
void parallelMergeSort(int *arr, int *temp, int left, int right, int threads) {
    if (threads <= 1 || right - left <= 1) {
        sort(arr + left, arr + right);
        return;
    }
    int mid = (left + right) / 2;

#pragma omp parallel sections
    {
#pragma omp section
        parallelMergeSort(arr, temp, left, mid, threads / 2);
#pragma omp section
        parallelMergeSort(arr, temp, mid, right, threads / 2);
    }

    int i = left, j = mid, k = left;
    while (i < mid && j < right) {
        if (arr[i] < arr[j]) temp[k++] = arr[i++];
        else temp[k++] = arr[j++];
    }
    while (i < mid) temp[k++] = arr[i++];
    while (j < right) temp[k++] = arr[j++];
    for (i = left; i < right; ++i) arr[i] = temp[i];
}

// kernel para calcular la máscara de bits en Radix Sort
__global__ void computeMaskKernel(int *d_input, int *d_mask, int bit, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        d_mask[idx] = (d_input[idx] >> bit) & 1;
    }
}

// kernel para calcular posiciones exclusivas de los 0s
__global__ void computeExclusiveScan(int *d_mask, int *d_scan, int n) {
    extern __shared__ int temp[];
    int idx = threadIdx.x;
    int i = blockIdx.x * blockDim.x + idx;

    if (i < n) {
        temp[idx] = d_mask[i];
    } else {
        temp[idx] = 0;
    }
    __syncthreads();

    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        int val = idx >= offset ? temp[idx - offset] : 0;
        __syncthreads();
        temp[idx] += val;
        __syncthreads();
    }

    if (i < n) {
        d_scan[i] = temp[idx];
    }
}

// kernel para realizar el reordenamiento (scatter)
__global__ void scatterKernel(int *d_input, int *d_output, int *d_mask, int *d_scan, int n, int totalZeros) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        int pos;
        if (d_mask[idx] == 0) {
            pos = d_scan[idx];
        } else {
            pos = totalZeros + idx - d_scan[idx];
        }
        d_output[pos] = d_input[idx];
    }
}

// fun Radix Sort en GPU
void radixSortGPU(int *arr, int n, int gridSize) {
    int *d_input, *d_output, *d_mask, *d_scan;
    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));
    hipMalloc(&d_mask, n * sizeof(int));
    hipMalloc(&d_scan, n * sizeof(int));

    hipMemcpy(d_input, arr, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    //dim3 gridSize((n + blockSize.x - 1) / blockSize.x); esta formula se uso para el benchmark (el primero) directo entre gpu vs cpu

    for (int bit = 0; bit < 32; ++bit) {
        computeMaskKernel<<<gridSize, blockSize>>>(d_input, d_mask, bit, n);
        hipDeviceSynchronize();

        computeExclusiveScan<<<gridSize, blockSize, blockSize.x * sizeof(int)>>>(d_mask, d_scan, n);
        hipDeviceSynchronize();

        int totalZeros;
        hipMemcpy(&totalZeros, &d_scan[n - 1], sizeof(int), hipMemcpyDeviceToHost);
        totalZeros += 1 - ((arr[n - 1] >> bit) & 1);

        scatterKernel<<<gridSize, blockSize>>>(d_input, d_output, d_mask, d_scan, n, totalZeros);
        hipDeviceSynchronize();

        swap(d_input, d_output);
    }

    hipMemcpy(arr, d_input, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
    hipFree(d_scan);
}

int main(int argc, char **argv) {
    if (argc != 4) {
        cerr << "Usage: ./prog <n> <mode> <threads_or_gridsize>" << endl;
        return 1;
    }

    int n = atoi(argv[1]);
    int mode = atoi(argv[2]);
    int threads_or_gridsize = atoi(argv[3]);

    int *arr = new int[n];
    int *temp = new int[n];
    //printf("Inicializando arreglo..."); fflush(stdout);
    initializeArray(arr, n);
    //printf("Arreglo inicializado\n");

    double startTime, endTime;

    const string csvFile = "../resultados/benchmark_results.csv";
    mkdir("../resultados", 0777);

    ofstream outFile(csvFile, ios::app);
    if (!outFile.is_open()) {
        cerr << "Error al abrir el archivo CSV para escribir: " << csvFile << endl;
        delete[] arr;
        delete[] temp;
        return 1;
    }
    //printf("Ordenando arreglo..."); fflush(stdout);
    if (mode == 0) { // CPU Mode
        omp_set_num_threads(threads_or_gridsize);
        startTime = omp_get_wtime();
        parallelMergeSort(arr, temp, 0, n, threads_or_gridsize);
        endTime = omp_get_wtime();
       // printf("Arreglo ordenado\n");
        printf("Tiempo MergeSort en CPU: %f segundos\n", (endTime - startTime)); fflush(stdout);
    } else if (mode == 1) { // GPU Mode
        startTime = omp_get_wtime();
        radixSortGPU(arr, n, threads_or_gridsize);
        endTime = omp_get_wtime();
       // printf("Arreglo ordenado\n");
        printf("Tiempo RadixSort en GPU: %f segundos\n", (endTime - startTime)); fflush(stdout);
    } else {
        cerr << "Invalid mode. Use 0 for CPU and 1 for GPU." << endl;
        delete[] arr;
        delete[] temp;
        return 1;
    }

    outFile << n << "," << mode << "," << threads_or_gridsize << "," << (endTime - startTime) << "\n";
    outFile.close();

    delete[] arr;
    delete[] temp;
    return 0;
}