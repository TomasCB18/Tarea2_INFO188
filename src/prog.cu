#include <iostream>
#include <vector>
#include <cstdlib>
#include <random> 
#include <omp.h>
#include <hip/hip_runtime.h>
#include <algorithm> 
#include <ctime>

using namespace std;

// Función para inicializar un arreglo con valores aleatorios
void initializeArray(int *arr, int n) {
    // Crear un generador de números aleatorios con la semilla basada en el tiempo actual
    std::mt19937 rng(static_cast<unsigned int>(std::time(0)));
    
    // Distribución uniforme en el rango [0, 1000000)
    std::uniform_int_distribution<int> dist(0, 999999);  
    
    for (int i = 0; i < n; ++i) {
        arr[i] = dist(rng);  // Genera un número aleatorio uniforme
    }
}
// Merge Sort paralelo para CPU
void parallelMergeSort(int *arr, int *temp, int left, int right, int threads) {
    if (threads <= 1 || right - left <= 1) {
        sort(arr + left, arr + right);
        return;
    }
    int mid = (left + right) / 2;

#pragma omp parallel sections
    {
#pragma omp section
        parallelMergeSort(arr, temp, left, mid, threads / 2);
#pragma omp section
        parallelMergeSort(arr, temp, mid, right, threads / 2);
    }

    // Merge
    int i = left, j = mid, k = left;
    while (i < mid && j < right) {
        if (arr[i] < arr[j]) temp[k++] = arr[i++];
        else temp[k++] = arr[j++];
    }
    while (i < mid) temp[k++] = arr[i++];
    while (j < right) temp[k++] = arr[j++];
    for (i = left; i < right; ++i) arr[i] = temp[i];
}

// Kernel para calcular la máscara de bits en Radix Sort
__global__ void computeMaskKernel(int *d_input, int *d_mask, int bit, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        d_mask[idx] = (d_input[idx] >> bit) & 1;
    }
}

// Kernel para calcular posiciones exclusivas de los 0s
__global__ void computeExclusiveScan(int *d_mask, int *d_scan, int n) {
    extern __shared__ int temp[];
    int idx = threadIdx.x;
    int i = blockIdx.x * blockDim.x + idx;

    if (i < n) {
        temp[idx] = d_mask[i];
    } else {
        temp[idx] = 0;
    }
    __syncthreads();

    // Exclusive scan (Blelloch)
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        int val = idx >= offset ? temp[idx - offset] : 0;
        __syncthreads();
        temp[idx] += val;
        __syncthreads();
    }

    if (i < n) {
        d_scan[i] = temp[idx];
    }
}

// Kernel para realizar el reordenamiento (scatter)
__global__ void scatterKernel(int *d_input, int *d_output, int *d_mask, int *d_scan, int n, int totalZeros) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        int pos;
        if (d_mask[idx] == 0) {
            pos = d_scan[idx];
        } else {
            pos = totalZeros + idx - d_scan[idx];
        }
        d_output[pos] = d_input[idx];
    }
}

// Función Radix Sort en GPU
void radixSortGPU(int *arr, int n) {
    int *d_input, *d_output, *d_mask, *d_scan;
    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));
    hipMalloc(&d_mask, n * sizeof(int));
    hipMalloc(&d_scan, n * sizeof(int));

    hipMemcpy(d_input, arr, n * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);

    for (int bit = 0; bit < 32; ++bit) {
        // Compute mask for the current bit
        computeMaskKernel<<<gridSize, blockSize>>>(d_input, d_mask, bit, n);
        hipDeviceSynchronize();

        // Compute exclusive scan
        computeExclusiveScan<<<gridSize, blockSize, blockSize.x * sizeof(int)>>>(d_mask, d_scan, n);
        hipDeviceSynchronize();

        // Compute totalZeros (last element of the scan array)
        int totalZeros;
        hipMemcpy(&totalZeros, &d_scan[n - 1], sizeof(int), hipMemcpyDeviceToHost);
        totalZeros += 1 - ((arr[n - 1] >> bit) & 1);

        // Scatter elements based on bit
        scatterKernel<<<gridSize, blockSize>>>(d_input, d_output, d_mask, d_scan, n, totalZeros);
        hipDeviceSynchronize();

        // Swap input and output arrays
        swap(d_input, d_output);
    }

    hipMemcpy(arr, d_input, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
    hipFree(d_scan);
}

int main(int argc, char **argv) {
    if (argc != 4) {
        cerr << "Usage: ./prog <n> <mode> <nt>" << endl;
        return 1;
    }

    int n = atoi(argv[1]);
    int mode = atoi(argv[2]);
    int threads = atoi(argv[3]);

    int *arr = new int[n];
    int *temp = new int[n];
    initializeArray(arr, n);

    double startTime, endTime;

    if (mode == 0) { // CPU Mode
        int *arrCopy = new int[n];
        copy(arr, arr + n, arrCopy);

        omp_set_num_threads(threads);
        startTime = omp_get_wtime();
        parallelMergeSort(arrCopy, temp, 0, n, threads);
        endTime = omp_get_wtime();
        cout << "CPU Merge Sort Time: " << (endTime - startTime) << " seconds" << endl;

        delete[] arrCopy;
    }

    else if (mode == 1) { // GPU Mode
        int *arrCopy = new int[n];
        copy(arr, arr + n, arrCopy);

        startTime = omp_get_wtime();
        radixSortGPU(arrCopy, n);
        endTime = omp_get_wtime();
        cout << "GPU Radix Sort Time: " << (endTime - startTime) << " seconds" << endl;

        delete[] arrCopy;
    }
    else {
        cerr << "Invalid mode. Use 0 for CPU and 1 for GPU." << endl;
        delete[] arr;
        delete[] temp;
        return 1;
    }

        delete[] arr;
        delete[] temp;
        return 0;
    }
